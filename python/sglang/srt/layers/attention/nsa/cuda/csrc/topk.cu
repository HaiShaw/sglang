#include "hip/hip_runtime.h"
#include <ATen/core/TensorBase.h>
#include <c10/cuda/CUDAStream.h>
#include <c10/macros/Macros.h>
#include <c10/util/Exception.h>
#include <cassert>
#include <cstddef>
#include <cstdint>
#include <optional>
#include <pybind11/pybind11.h>
#include <pybind11/stl.h>
#include <torch/python.h>

namespace {

constexpr int TopK = 2048;
constexpr int kThreadsPerBlock = 1024;
constexpr size_t kSmem = 32 * 1024 * sizeof(uint32_t); // 128KB

struct FastTopKParams {
  const float *__restrict__ input; // [B, input_stride]
  int32_t *__restrict__ indices;   // [B, TopK]
  int32_t *__restrict__ lengths;   // [B]
  int64_t input_stride;
  bool use_tilelang;
};

// when length <= TopK, we can directly write the indices
__device__ void naive_topk_cuda(const float *__restrict__ score,
                                int32_t *__restrict__ indice, int32_t length) {
  const auto tid = threadIdx.x;
  for (int i = tid; i < TopK; i += kThreadsPerBlock) {
    indice[i] = (i < length) ? i : -1;
  }
}

// keep the first `length` entries, set others to -1
__device__ void
naive_topk_transform(const float *__restrict__ score, int32_t length,
                     int32_t *__restrict__ dst_page_table,
                     const int32_t *__restrict__ src_page_table) {
  const auto tid = threadIdx.x;
  for (auto i = tid; i < TopK; i += kThreadsPerBlock) {
    dst_page_table[i] = (i < length) ? src_page_table[i] : -1;
  }
}

__device__ __forceinline__ uint8_t convert_to_uint8(float x) {
  __half h = __float2half_rn(x);
  uint16_t bits = __half_as_ushort(h);
  uint16_t key = (bits & 0x8000) ? static_cast<uint16_t>(~bits & 0xFFFF)
                                 : static_cast<uint16_t>(bits | 0x8000);
  return static_cast<uint8_t>(key >> 8);
}

__device__ __forceinline__ uint32_t convert_to_uint32(float x) {
  uint32_t bits = __float_as_uint(x);
  return (bits & 0x80000000u) ? (~bits & 0xFFFFFFFFu) : (bits | 0x80000000u);
}

template <bool Is_Epilogue = false, typename Indexer, typename Loader,
          int LENGTH, int MAX_REMAIN>
__device__ __forceinline__ auto
radix_topk(Indexer indexer, Loader loader, uint32_t length, int topk,
           int *__restrict__ index, int &__restrict__ s_counter,
           int (&__restrict__ s_histogram)[LENGTH],
           int &__restrict__ s_remain_cnt,
           int (&__restrict__ s_remain_idx)[MAX_REMAIN]) -> int {
  constexpr auto RADIX = LENGTH - 1;
  static_assert(RADIX > 1 && (RADIX & (RADIX - 1)) == 0,
                "RADIX must be power of 2");
  static_assert(RADIX <= kThreadsPerBlock);
  __shared__ uint32_t s_threshold_bin_id;

  const auto tx = threadIdx.x;
  if (tx < RADIX + 1)
    s_histogram[tx] = 0;
  __syncthreads();

  /// NOTE: Use uint32_t as the index
  for (auto i = tx; i < length; i += kThreadsPerBlock) {
    const auto idx = indexer(i);
    const auto bin = loader(idx);
    ::atomicAdd(&s_histogram[bin], 1);
  }
  __syncthreads();

  // cumsum (descending)
  if (tx == 0) {
    s_histogram[RADIX] = 0;
    s_remain_cnt = 0;
    for (int i = RADIX - 2; i >= 0; --i) {
      s_histogram[i] += s_histogram[i + 1];
    }
    // threshold bin
    for (int i = 0; i < RADIX; i++) {
      if (s_histogram[i] >= topk && s_histogram[i + 1] < topk) {
        s_threshold_bin_id = i;
        break;
      }
    }
  }
  __syncthreads();

  const auto threshold_bin = s_threshold_bin_id;
  const auto new_topk = topk - s_histogram[threshold_bin + 1];

  for (auto i = tx; i < length; i += kThreadsPerBlock) {
    const auto idx = indexer(i);
    const auto bin_id = static_cast<uint32_t>(loader(idx));
    if (bin_id > threshold_bin) {
      index[::atomicAdd(&s_counter, 1)] = idx;
    } else if (bin_id == threshold_bin && new_topk > 0) {
      if constexpr (Is_Epilogue) {
        index[::atomicAdd(&s_counter, 1)] = idx;
      } else {
        if (const auto cnt = ::atomicAdd(&s_remain_cnt, 1);
            C10_LIKELY(cnt < MAX_REMAIN)) {
          s_remain_idx[cnt] = idx;
        }
      }
    }
  }
  __syncthreads();

  return new_topk;
}

__device__ void fast_topk_cuda(const float *__restrict__ input,
                               int *__restrict__ index, int length,
                               int topk = TopK) {
  constexpr auto RADIX = 256;
  constexpr auto SMEM_INPUT_SIZE = kSmem / (2 * sizeof(int));

  __shared__ int s_histogram[RADIX + 1];
  __shared__ int s_num_input[2];
  __shared__ int s_counter;

  // allocate for two rounds
  extern __shared__ int s_input_idx[][SMEM_INPUT_SIZE];
  s_counter = 0;

  // collect candidates
  const auto indexer = [](int idx) { return idx; };
  const auto loader = [&input](int idx) {
    return convert_to_uint8(input[idx]);
  };
  int new_topk = radix_topk(indexer, loader, length, topk, index, s_counter,
                            s_histogram, s_num_input[0], s_input_idx[0]);
  if (new_topk <= 0)
    return;

  // round 0
  const auto indexer_0 = [](int idx) { return s_input_idx[0][idx]; };
  const auto loader_0 = [&input](int idx) {
    return (convert_to_uint32(input[idx]) >> 24) & 0xFF;
  };
  new_topk = radix_topk(indexer_0, loader_0, s_num_input[0], new_topk, index,
                        s_counter, s_histogram, s_num_input[1], s_input_idx[1]);
  if (new_topk <= 0)
    return;

  // round 1
  const auto indexer_1 = [](int idx) { return s_input_idx[1][idx]; };
  const auto loader_1 = [&input](int idx) {
    return (convert_to_uint32(input[idx]) >> 16) & 0xFF;
  };
  new_topk = radix_topk(indexer_1, loader_1, s_num_input[1], new_topk, index,
                        s_counter, s_histogram, s_num_input[0], s_input_idx[0]);
  if (new_topk <= 0)
    return;

  // round 2
  const auto loader_2 = [&input](int idx) {
    return (convert_to_uint32(input[idx]) >> 8) & 0xFF;
  };
  new_topk = radix_topk(indexer_0, loader_2, s_num_input[0], new_topk, index,
                        s_counter, s_histogram, s_num_input[1], s_input_idx[1]);
  if (new_topk <= 0)
    return;

  // round 3
  const auto loader_3 = [&input](int idx) {
    return convert_to_uint32(input[idx]) & 0xFF;
  };
  // epilogue
  radix_topk<true>(indexer_1, loader_3, s_num_input[1], new_topk, index,
                   s_counter, s_histogram, s_num_input[0], s_input_idx[0]);
}

__device__ void fast_topk_cuda_tl(const float *__restrict__ input,
                                  int *__restrict__ index, int length,
                                  int topk = TopK) {
  constexpr auto BLOCK_SIZE = 1024;
  constexpr auto RADIX = 256;
  constexpr auto SMEM_INPUT_SIZE = kSmem / (2 * sizeof(int));

  __shared__ int s_threshold_bin_id;
  __shared__ int s_histogram[RADIX + 1];
  __shared__ int s_num_input[2];
  __shared__ int s_counter;

  // allocate for two rounds
  extern __shared__ int s_input_idx[][SMEM_INPUT_SIZE];

  int tx = threadIdx.x;

  // stage 1: 8bit coarse histogram
  if (tx < RADIX + 1)
    s_histogram[tx] = 0;
  __syncthreads();

  for (int idx = tx; idx < length; idx += BLOCK_SIZE) {
    const auto bin = convert_to_uint8(input[idx]);
    ::atomicAdd(&s_histogram[bin], 1);
  }
  __syncthreads();

  // cumsum (descending)
  if (tx == 0) {
    for (int i = RADIX - 2; i >= 0; --i) {
      s_histogram[i] += s_histogram[i + 1];
    }
    // threshold bin
    for (int i = 0; i < RADIX; i++) {
      if (s_histogram[i] >= topk && s_histogram[i + 1] < topk) {
        s_threshold_bin_id = i;
        break;
      }
    }
    s_num_input[0] = 0;
    s_counter = 0;
  }
  __syncthreads();

  int threshold_bin = s_threshold_bin_id;
  int new_topk = topk - s_histogram[threshold_bin + 1];

  // collect candidates
  for (int idx = tx; idx < length; idx += BLOCK_SIZE) {
    const auto bin_id = static_cast<int>(convert_to_uint8(input[idx]));
    if (bin_id > threshold_bin) {
      int pos = ::atomicAdd(&s_counter, 1);
      index[pos] = idx;
    } else if (bin_id == threshold_bin && new_topk > 0) {
      int pos = ::atomicAdd(&s_num_input[0], 1);
      if (pos < SMEM_INPUT_SIZE) {
        [[likely]] s_input_idx[0][pos] = idx;
      }
    }
  }
  __syncthreads();

  // stage 2: refine with 8bit radix passes
#pragma unroll 4
  for (int round = 0; round < 4; ++round) {
    if (new_topk <= 0)
      break;
    int r_idx = round % 2;

    // reset
    if (tx < RADIX + 1)
      s_histogram[tx] = 0;
    __syncthreads();

    int num_input = s_num_input[r_idx];
    for (int i = tx; i < num_input; i += BLOCK_SIZE) {
      int idx = s_input_idx[r_idx][i];
      uint32_t bin32 =
          (convert_to_uint32(input[idx]) >> (24 - round * 8)) & 0xFF;
      ::atomicAdd(&s_histogram[bin32], 1);
    }
    __syncthreads();

    if (tx == 0) {
      for (int i = RADIX - 2; i >= 0; --i)
        s_histogram[i] += s_histogram[i + 1];
      for (int i = 0; i < RADIX; i++) {
        if (s_histogram[i] >= new_topk && s_histogram[i + 1] < new_topk) {
          s_threshold_bin_id = i;
          break;
        }
      }
      s_num_input[r_idx ^ 1] = 0;
    }
    __syncthreads();

    new_topk -= s_histogram[s_threshold_bin_id + 1];
    int threshold_bin = s_threshold_bin_id;

    for (int i = tx; i < num_input; i += BLOCK_SIZE) {
      int idx = s_input_idx[r_idx][i];
      uint32_t bin32 =
          (convert_to_uint32(input[idx]) >> (24 - round * 8)) & 0xFF;
      if (bin32 > threshold_bin) {
        int pos = ::atomicAdd(&s_counter, 1);
        index[pos] = idx;
      } else if (bin32 == threshold_bin && new_topk > 0) {
        if (round == 3) {
          int pos = ::atomicAdd(&s_counter, 1);
          index[pos] = idx;
        } else {
          int pos = ::atomicAdd(&s_num_input[r_idx ^ 1], 1);
          if (pos < SMEM_INPUT_SIZE)
            s_input_idx[r_idx ^ 1][pos] = idx;
        }
      }
    }
    __syncthreads();
  }
}

__global__ void topk_kernel(const FastTopKParams params) {
  const auto &[input, indices, lengths, input_stride, use_tilelang] = params;
  const auto bid = blockIdx.x;
  const auto length = *(lengths + bid);
  const auto indice = indices + bid * TopK;
  const auto score = input + bid * input_stride;
  if (length <= TopK) {
    return naive_topk_cuda(score, indice, length);
  } else {
    if (use_tilelang) {
      return fast_topk_cuda_tl(score, indice, length);
    } else {
      return fast_topk_cuda(score, indice, length);
    }
  }
}

__global__ void topk_kernel_transform_decode( // decode
    const FastTopKParams params, int32_t *__restrict__ dst_page_table,
    const int32_t *__restrict__ src_page_table, const int64_t src_stride) {
  const auto &[input, _, lengths, input_stride, use_tilelang] = params;
  const auto bid = blockIdx.x;
  const auto tid = threadIdx.x;
  const auto length = *(lengths + bid);
  const auto src_page_entry = src_page_table + bid * src_stride;
  const auto dst_page_entry = dst_page_table + bid * TopK;
  const auto score = input + bid * input_stride;
  if (length <= TopK) {
    return naive_topk_transform(score, length, dst_page_entry, src_page_entry);
  } else {
    __shared__ int s_indices[TopK];
    if (use_tilelang) {
      fast_topk_cuda_tl(score, s_indices, length);
    } else {
      fast_topk_cuda(score, s_indices, length);
    }
    // copy src[s_indices] to dst, we manually unroll here
    static_assert(TopK % kThreadsPerBlock == 0);
    static_assert(TopK / kThreadsPerBlock == 2);
    const auto idx_0 = tid;
    const auto pos_0 = s_indices[idx_0];
    dst_page_entry[idx_0] = src_page_entry[pos_0];
    const auto idx_1 = tid + kThreadsPerBlock;
    const auto pos_1 = s_indices[idx_1];
    dst_page_entry[idx_1] = src_page_entry[pos_1];
  }
}

__global__ void topk_kernel_transform_prefill( // prefill
    const FastTopKParams params, int32_t *__restrict__ dst_page_table,
    const int32_t *__restrict__ src_page_table, const int64_t src_stride,
    const int32_t *__restrict__ cu_seqlens, const int64_t prefill_bs) {
  const auto &[input, _, lengths, input_stride, use_tilelang] = params;
  const auto bid = blockIdx.x;
  const auto tid = threadIdx.x;
  const auto length = *(lengths + bid);
  const auto dst_page_entry = dst_page_table + bid * TopK;
  const auto score = input + bid * input_stride;

  /// NOTE: prefill bs is usually small, we can just use a simple loop here
  /// We ensure that last cu_seqlens is equal to number of blocks launched
  assert(gridDim.x == cu_seqlens[prefill_bs] &&
         "Invalid cu_seqlens in topk-transform-prefill");
  __shared__ const int32_t *s_src_page_entry;
  if (tid == 0) {
    for (int64_t offset = 0; offset < prefill_bs; ++offset) {
      if (bid < cu_seqlens[offset + 1]) {
        s_src_page_entry = src_page_table + offset * src_stride;
        break;
      }
    }
  }
  __syncthreads();
  const auto src_page_entry = s_src_page_entry;

  if (length <= TopK) {
    return naive_topk_transform(score, length, dst_page_entry, src_page_entry);
  } else {
    __shared__ int s_indices[TopK];
    if (use_tilelang) {
      fast_topk_cuda_tl(score, s_indices, length);
    } else {
      fast_topk_cuda(score, s_indices, length);
    }
    // copy src[s_indices] to dst, we manually unroll here
    static_assert(TopK % kThreadsPerBlock == 0);
    static_assert(TopK / kThreadsPerBlock == 2);
    const auto idx_0 = tid;
    const auto pos_0 = s_indices[idx_0];
    dst_page_entry[idx_0] = src_page_entry[pos_0];
    const auto idx_1 = tid + kThreadsPerBlock;
    const auto pos_1 = s_indices[idx_1];
    dst_page_entry[idx_1] = src_page_entry[pos_1];
  }
}

auto get_params(at::Tensor score, at::Tensor lengths, bool use_tilelang,
                std::optional<at::Tensor> indices_opt = std::nullopt)
    -> FastTopKParams {
  const auto B = score.size(0);
  TORCH_CHECK(score.dim() == 2 && score.stride(1) == 1);
  TORCH_CHECK(lengths.dim() == 1 && lengths.is_contiguous());
  TORCH_CHECK(lengths.size(0) == B);
  int32_t *indices_data_ptr = nullptr;
  if (indices_opt.has_value()) {
    const auto &indices = indices_opt.value();
    TORCH_CHECK(indices.dim() == 2 && indices.is_contiguous());
    TORCH_CHECK(indices.size(0) == B);
    TORCH_CHECK(indices.size(1) == TopK);
    indices_data_ptr = indices.data_ptr<int32_t>();
  }

  return FastTopKParams{
      .input = score.data_ptr<float>(),
      .indices = indices_data_ptr,
      .lengths = lengths.data_ptr<int32_t>(),
      .input_stride = score.stride(0),
      .use_tilelang = use_tilelang,
  };
}

template <auto *f, size_t max_dynamic_smem>
auto setup_kernel_smem_once() -> void {
  [[maybe_unused]]
  static const auto result = [] {
    return ::hipFuncSetAttribute(reinterpret_cast<const void*>(
        f), ::hipFuncAttributeMaxDynamicSharedMemorySize, max_dynamic_smem);
  }();
  TORCH_CHECK(result == hipSuccess,
              "set_up_kernel_once failed:", ::hipGetErrorString(result));
}

auto fast_topk_interface(at::Tensor score, at::Tensor indices,
                         at::Tensor lengths, bool use_tilelang) -> void {
  const auto params = get_params(score, lengths, use_tilelang, indices);
  const auto B = score.size(0);
  const auto stream = at::cuda::getCurrentCUDAStream().stream();
  const auto grid = dim3{static_cast<uint32_t>(B)};
  const auto block = dim3{kThreadsPerBlock};
  setup_kernel_smem_once<topk_kernel, kSmem>();
  topk_kernel<<<grid, block, kSmem, stream>>>(params);
  const auto result = hipGetLastError();
  TORCH_CHECK(result == hipSuccess,
              "topk kernel failed:", ::hipGetErrorString(result));
}

auto fast_topk_transform_interface(at::Tensor score, at::Tensor lengths,
                                   at::Tensor dst_page_table,
                                   at::Tensor src_page_table,
                                   at::Tensor cu_seqlens,
                                   bool use_tilelang) -> void {
  const auto params = get_params(score, lengths, use_tilelang);
  const auto B = score.size(0);
  TORCH_CHECK(dst_page_table.dim() == 2 && dst_page_table.is_contiguous());
  TORCH_CHECK(src_page_table.dim() == 2 && src_page_table.stride(1) == 1);
  TORCH_CHECK(cu_seqlens.dim() == 1 && cu_seqlens.is_contiguous());
  const auto prefill_bs = cu_seqlens.size(0) - 1;
  TORCH_CHECK(dst_page_table.size(0) == B);
  TORCH_CHECK(dst_page_table.size(1) == TopK);
  TORCH_CHECK(src_page_table.size(0) == prefill_bs);
  TORCH_CHECK(prefill_bs <= B); // prefill_bs should be smaller than expanded bs

  // launch kernel
  const auto stream = at::cuda::getCurrentCUDAStream().stream();
  const auto grid = dim3{static_cast<uint32_t>(B)};
  const auto block = dim3{kThreadsPerBlock};
  const auto src_stride = src_page_table.stride(0);

  // dispatch to decode or prefill
  const auto is_decode = (prefill_bs == B);
  if (is_decode) {
    setup_kernel_smem_once<topk_kernel_transform_decode, kSmem>();
    topk_kernel_transform_decode<<<grid, block, kSmem, stream>>>(
        params, dst_page_table.data_ptr<int32_t>(),
        src_page_table.data_ptr<int32_t>(), src_stride);
  } else {
    setup_kernel_smem_once<topk_kernel_transform_prefill, kSmem>();
    topk_kernel_transform_prefill<<<grid, block, kSmem, stream>>>(
        params, dst_page_table.data_ptr<int32_t>(),
        src_page_table.data_ptr<int32_t>(), src_stride,
        cu_seqlens.data_ptr<int32_t>(), prefill_bs);
  }

  const auto result = hipGetLastError();
  TORCH_CHECK(result == hipSuccess,
              "topk kernel failed:", ::hipGetErrorString(result));
}

} // namespace

PYBIND11_MODULE(topk_kernel, m) {
  m.def("fast_topk", &fast_topk_interface);
  m.def("fast_topk_transform", &fast_topk_transform_interface);
}
